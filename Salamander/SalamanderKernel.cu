#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <cstdlib>  // For malloc, free, and rand
#include <ctime>    // For time (used for srand)

constexpr int NUM_USERS = 1000;
constexpr int NUM_ITEMS = 1000;
constexpr int K = 10; // Number of latent features
constexpr int NUM_EPOCHS = 20;
constexpr float LEARNING_RATE = 0.01f;
constexpr float LAMBDA = 0.02f; // Regularization parameter

// CUDA kernel for matrix factorization
__global__ void matrixFactorizationSGD(
    int* userIds, int* itemIds, float* ratings,
    float* P, float* Q, int numInteractions) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numInteractions) return;

    int u = userIds[idx];
    int i = itemIds[idx];
    float r = ratings[idx];

    float* Pu = &P[u * K];
    float* Qi = &Q[i * K];

    // Compute the dot product for predicted rating
    float pred = 0.0f;
    for (int k = 0; k < K; ++k) {
        pred += Pu[k] * Qi[k];
    }

    // Compute error
    float err = r - pred;

    // Update P and Q matrices
    for (int k = 0; k < K; ++k) {
        float gradPu = err * Qi[k] - LAMBDA * Pu[k];
        float gradQi = err * Pu[k] - LAMBDA * Qi[k];

        Pu[k] += LEARNING_RATE * gradPu;
        Qi[k] += LEARNING_RATE * gradQi;
    }
}

// Utility function for random data generation (host-side)
void generateData(int numUsers, int numItems, int numInteractions,
    int* userIds, int* itemIds, float* ratings) {
    srand(time(0));
    for (int i = 0; i < numInteractions; ++i) {
        userIds[i] = rand() % numUsers;
        itemIds[i] = rand() % numItems;
        ratings[i] = static_cast<float>(rand() % 5 + 1); // Ratings between 1 and 5
    }
}

// Exposed function for running the CUDA kernel
extern "C" void runMatrixFactorization(
    int* userIds, int* itemIds, float* ratings,
    float* P, float* Q,
    int numUsers, int numItems, int numInteractions, int numEpochs) {

    // Allocate device memory
    int* d_userIds, * d_itemIds;
    float* d_ratings, * d_P, * d_Q;

    hipMalloc(&d_userIds, numInteractions * sizeof(int));
    hipMalloc(&d_itemIds, numInteractions * sizeof(int));
    hipMalloc(&d_ratings, numInteractions * sizeof(float));
    hipMalloc(&d_P, numUsers * K * sizeof(float));
    hipMalloc(&d_Q, numItems * K * sizeof(float));

    hipMemcpy(d_userIds, userIds, numInteractions * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_itemIds, itemIds, numInteractions * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_ratings, ratings, numInteractions * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_P, P, numUsers * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Q, Q, numItems * K * sizeof(float), hipMemcpyHostToDevice);

    // Launch CUDA kernel
    int blockSize = 256;
    int gridSize = (numInteractions + blockSize - 1) / blockSize;

    for (int epoch = 0; epoch < numEpochs; ++epoch) {
        matrixFactorizationSGD << <gridSize, blockSize >> > (d_userIds, d_itemIds, d_ratings, d_P, d_Q, numInteractions);
        hipDeviceSynchronize();
    }

    // Copy results back to host
    hipMemcpy(P, d_P, numUsers * K * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Q, d_Q, numItems * K * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_userIds);
    hipFree(d_itemIds);
    hipFree(d_ratings);
    hipFree(d_P);
    hipFree(d_Q);
}
